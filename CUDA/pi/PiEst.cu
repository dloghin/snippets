/* Pi estimation using Monte Carlo simulation
 * implemented using XorShift rng
 *
 * Dumi Loghin, 2015-2017
 *
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <sys/time.h>

typedef float sFloat;
typedef int64_t s64Int;

#define MaxCudaBlocks			2
#define MaxCudaThreadsBlock 		512
//#define MaxCudaBlocks			8
//#define MaxCudaThreadsBlock 		1024


#define SIZE 100000

#define MAX64 0x7FFFFFFFFFFFFFFF

__host__ __device__ s64Int xorshift64star(s64Int x) {
	x ^= x >> 12; 	// a
	x ^= x << 25; 	// b
	x ^= x >> 27; 	// c
	return x * UINT64_C(2685821657736338717);
}

__global__ void pikernel(s64Int samples, s64Int* nIn, s64Int* nOut) {
	s64Int i;	
	sFloat xx, yy;
	s64Int ni = 0;	// points inside the circle
	s64Int no = 0;	// points outside the circle

	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	// initialize the random generator (non-zero)
	s64Int offset = tid * samples;
	s64Int x = 12345678 + offset * 89482311;
	s64Int y = 87654321 + offset + 12345678;

	// generate points and count
	for(i = 0; i < samples; i++) {
		x = xorshift64star(x);
		y = xorshift64star(y);
		xx = ((sFloat)x)/(sFloat)MAX64;
		yy = ((sFloat)y)/(sFloat)MAX64;
		if (xx * xx + yy * yy > 1.0)
			no++;
		else
			ni++;
	}

	// update output
	nIn[tid] = ni;
	nOut[tid] = no;
}

unsigned long long int get_time() {
        struct timeval tv;
        gettimeofday(&tv, NULL);
        return (unsigned long long int)(tv.tv_usec + tv.tv_sec * 1000000);
}

int main(int argc, char** argv) {
	int src_line;
	hipError_t rc;
	s64Int size = SIZE, i, sIn = 0, sOut = 0;
	s64Int *dIn, *dOut;
	long long int t1, t2;

	printf("Using %d blocks and %d threads per block\n", MaxCudaBlocks, MaxCudaThreadsBlock);

	// alloc memory using unified memory
	src_line = __LINE__;
	rc = hipMallocManaged((void **)&dIn, MaxCudaBlocks * MaxCudaThreadsBlock * sizeof(s64Int));
	if (rc != hipSuccess)
		goto lbl_err;
	src_line = __LINE__;
	rc = hipMallocManaged((void **)&dOut, MaxCudaBlocks * MaxCudaThreadsBlock * sizeof(s64Int));
	if (rc != hipSuccess)
		goto lbl_err;

	// call kernel
	t1 = get_time();
	pikernel<<<MaxCudaBlocks,MaxCudaThreadsBlock>>>(size, dIn, dOut);
	src_line = __LINE__;
	rc = hipDeviceSynchronize();
	t2 = get_time();
	if (rc != hipSuccess)
                goto lbl_err;
	
	// compute pi
	for (i = 0; i < MaxCudaBlocks * MaxCudaThreadsBlock; i++) {
		sIn += dIn[i];
		sOut += dOut[i];
	}

	hipFree(dIn);
	hipFree(dOut);

	printf("Estimated value of Pi is %3.6f\n", 4.0 * sIn / (sIn + sOut));
	printf("Estimation took %lld usec\n", t2-t1);

	return 0;

lbl_err:
    	printf("Launch CUDA error %s al line %d\n", hipGetErrorString(rc), src_line);
	return -1;
}
